#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"

#include <stdio.h>

#include "coeff.h"
#include "range.h"

using namespace std;

typedef enum
{
    GetTorqueAngvelPair,
    GetOneEfficiencyValue,
    GenerateMap
} operation;

typedef struct
{
    int m;
    float theta;
} dynamics;

typedef struct
{
    float Mdemand, Omega;
} motor;

//TODO: new func for dynamics and motor

typedef coeff_Rslt* coeff;

__device__ float calculate_Mdemand(dynamics* d, coeff c, float v, float i)
{
    const float h0 = ((((c->c[4] * d->theta + c->c[3]) * d->theta + c->c[2]) * d->theta + c->c[1]) * d->theta + c->c[0]);
    return (d->m * h0 + c->cF * v * v) / i;
}

__device__ float calculate_Omega(float v, coeff c)
{
    return v * c->cOmega;
}


/*__global__ void calcValues(float* p_values, float* m_values, const float* c_values, const float m, const float c_F, const float r_dk, const float i_gbx) {
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    const int zIndex = blockIdx.z * blockDim.z + threadIdx.z;

    const float theta = xIndex * 0.1f;
    const float v = yIndex * 0.1f;
    const float i = zIndex * 0.1f;

    

    const int index = xIndex + yIndex * blockDim.x + zIndex * blockDim.x * blockDim.y;
    p_values[index] = h0;
    m_values[index] = Omega;
}*/

__global__ void calculate_values(float* output_mdemand, float* output_omega, float* output_color, float theta_step, float v_step, float igbx, float mdemand_coeff, float cf_coeff, float rd_k)
{
    // Calculate the thread's x, y, and z indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate the corresponding values of theta and v based on the indices
    float theta = idx * theta_step;
    float v = idy * v_step;


}

coeff_Rslt* creatCoeffs()
{
    coeff_Road* cRoad = newCoeff(9.8067, 0.01);
    coeff_Aero* cAero = newCoeff(1.2041, 1.05, 4);
    coeff_Trac* cTrac = newCoeff(4, 0.9, 0.98, 0.3);

    coeff_Rslt* cRslt = calcCoeffs(cRoad, cAero, cTrac);

    free(cRoad);
    free(cAero);
    free(cTrac);
}

int main() {
    //Prepare
    coeff_Rslt* coeffs = creatCoeffs();

    range
        * iRange = newRange(0, 33, 0.1f),
        * vRange = newRange(0, 33, 0.1f);
    
    float* Mdemand, d_Mdemand;
    float* Omega, d_Omega;

    operation op = GetTorqueAngvelPair;

    switch (op)
    {
    case GetTorqueAngvelPair:

        //Device alloc
        const int size = sizeFromRange(iRange);
        const int numValues = size * size * size;

        hipMallocManaged((void**)&d_Mdemand, numValues * sizeof(float));
        hipMallocManaged((void**)&d_Omega, numValues * sizeof(float));

        //Input upload
        a = 2;
        b = 7;

        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

        //Perform calculations on device
        dim3 threadsPerBlock(8, 8, 8);
        dim3 numBlocks(size / threadsPerBlock.x, size / threadsPerBlock.y, size / threadsPerBlock.z);

        calcValues << <numBlocks, threadsPerBlock >> > (p_values, m_values, m, c_F, r_dk, i_gbx);

        hipDeviceSynchronize();

        //Output download
        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        //Cleanup on device
        hipFree(p_values);
        hipFree(m_values);
        break;
    case GetOneEfficiencyValue:
        break;
    case GenerateMap:
        break;
    }

    //Cleanup on host
    free(pointers);
    return 0;
}
